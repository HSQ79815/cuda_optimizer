
#include <hip/hip_runtime.h>
#include <cmath>
#include <cstdint>
#include <hip/hip_runtime_api.h>
#include <iostream>
#include <sstream>
#include <string>
#include <vector>

#define DivUp(x, y) (x + y - 1) / y

template <typename T, size_t THREADS_PER_BLOCK> __global__ void Reduce(const T* src, T* dest, size_t N)
{
    __shared__ T sdata[THREADS_PER_BLOCK];
    unsigned int tid = threadIdx.x;
    unsigned int idx = tid + blockDim.x * blockIdx.x;

    sdata[tid] = idx < N ? src[idx] : T{};
    __syncthreads();

    for (unsigned int stride = THREADS_PER_BLOCK / 2; stride > 0; stride >>= 1) {
        if (tid < stride) {
            sdata[tid] += sdata[tid + stride];
        }
        __syncthreads();
    }
    if (tid == 0) {
        dest[blockIdx.x] = sdata[0];
    }
}

template <typename T> bool Check(const T* s1, const T* s2, size_t N, float epsilon)
{
    for (size_t i = 0; i < N; ++i) {
        if (abs(s1[i] - s2[i]) > epsilon) {
            return false;
        }
    }
    return true;
}

template <typename T, size_t THREADS_PER_BLOCK> void ReduceCpu(const T* src, T* dest, size_t N)
{
    size_t num = N / THREADS_PER_BLOCK;
    size_t i   = 0;
    for (; i < num; ++i) {
        T      t        = T{};
        size_t base_idx = i * THREADS_PER_BLOCK;
        for (size_t j = 0; j < THREADS_PER_BLOCK; ++j) {
            t += src[base_idx + j];
        }
        dest[i] = t;
    }
    if (N % THREADS_PER_BLOCK != 0) {
        T& t = dest[i];
        t    = T{};
        for (size_t j = num * THREADS_PER_BLOCK; j < N; ++j) {
            t += src[j];
        }
    }
}

int main(int argc, char** argv)
{
    using type         = float;
    constexpr size_t N = 128 * 1024 * 1024;

    constexpr size_t THREADS_PER_BLOCK = 256;

    constexpr size_t  dest_N = DivUp(N, THREADS_PER_BLOCK);
    std::vector<type> cpu_src(N, 1.02);
    std::vector<type> cpu_dest(dest_N);

    ReduceCpu<type, THREADS_PER_BLOCK>(&cpu_src[0], &cpu_dest[0], N);

    type* device_src  = nullptr;
    type* device_dest = nullptr;

    std::vector<type> device_to_cpu(dest_N);

    hipMalloc(&device_src, N * sizeof(type));
    hipMalloc(&device_dest, dest_N * sizeof(type));

    hipMemcpy(device_src, &cpu_src[0], N * sizeof(type), hipMemcpyHostToDevice);

    dim3 block(THREADS_PER_BLOCK, 1, 1);
    dim3 grid(DivUp(N, THREADS_PER_BLOCK), 1, 1);
    Reduce<type, THREADS_PER_BLOCK><<<grid, block>>>(device_src, device_dest, N);

    float       elapsedTime;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start, 0);
    Reduce<type, THREADS_PER_BLOCK><<<grid, block>>>(device_src, device_dest, N);
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsedTime, start, stop);
    hipMemcpy(&device_to_cpu[0], device_dest, dest_N * sizeof(type), hipMemcpyDeviceToHost);

    float epsilon = 1e-5;
    if (!Check(&cpu_dest[0], &device_to_cpu[0], dest_N, epsilon)) {
        std::cout << "result wrong!\n";
        std::cout << cpu_dest[0] << "\t" << cpu_dest[1] << "\t" << cpu_dest[2] << "\t" << cpu_dest[3] << "\t"
                  << cpu_dest[4] << "\t" << cpu_dest[5] << "\n";
        std::cout << device_to_cpu[0] << "\t" << device_to_cpu[1] << "\t" << device_to_cpu[2] << "\t"
                  << device_to_cpu[3] << "\t" << device_to_cpu[4] << "\t" << device_to_cpu[5] << "\n";
    }

    std::cout << "average cost: " << elapsedTime << " ms\n";
    float bandwidth = (N + dest_N) * sizeof(type) / elapsedTime * 1000 / (1 << 30);
    std::cout << "bandwidth: " << bandwidth << " GB/s\n";

    hipFree(device_src);
    hipFree(device_dest);
    hipEventDestroy(start);
    hipEventDestroy(stop);

    return 0;
}
