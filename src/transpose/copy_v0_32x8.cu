#include "hip/hip_runtime.h"

#include <cmath>
#include <cstdint>
#include <hip/hip_runtime_api.h>
#include <gflags/gflags.h>
#include <glog/logging.h>
#include <iomanip>
#include <iostream>
#include <sstream>
#include <string>
#include <vector>

#define DivUp(x, y) (x + y - 1) / y

template <typename T> void __global__ MatrixCopy0(const T* src, int M, int N, T* dst)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < N && y < M) {
        int idx = y * N + x;
        dst[idx] = src[idx];
    }
}

int main (int argc, char** argv){
    gflags::ParseCommandLineFlags(&argc, &argv, true);
    google::InitGoogleLogging(argv[0]);
    google::LogToStderr();

    using type         = float;

    constexpr int M = 1 << 13;
    constexpr int N = 1 << 13;

    constexpr int size_in_bytes = M * N * sizeof(type);

    std::vector<type> cpu_src(M*N, type(1));

    type* device_src  = nullptr;
    type* device_dest = nullptr;

    hipMalloc(&device_src, size_in_bytes);
    hipMalloc(&device_dest, size_in_bytes);

    hipMemcpy(device_src, &cpu_src[0],size_in_bytes, hipMemcpyHostToDevice);

    dim3 block(32, 8, 1);
    dim3 grid(DivUp(M, 32), DivUp(N, 8), 1);
    MatrixCopy0<<<grid,block>>>(device_src,M,N,device_dst);

    float       elapsedTime;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);
    MatrixCopy0<<<grid,block>>>(device_src,M,N,device_dst);
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsedTime, start, stop);
    hipMemcpy(&device_to_cpu[0], device_dest, dest_N * sizeof(type), hipMemcpyDeviceToHost);

    LOG(INFO) << "average cost: " << elapsedTime << " ms";
    float bandwidth = size_in_bytes * 2 / elapsedTime * 1000 / (1 << 30);
    LOG(INFO) << "bandwidth: " << bandwidth << " GB/s";

    hipFree(device_src);
    hipFree(device_dest);
    hipEventDestroy(start);
    hipEventDestroy(stop);

    return 0;
}
